#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <assert.h>

using namespace std;

vector<int> seqMatmul(vector<int> A, vector<int> B, int M) {
	vector<int> C(A.size());

	int i,j,k;
	for (i = 0; i < M; i++) {
		for (j = 0; j < M; j++) {
			for (k = 0; k < M; k++) {
				C[i*M + j] += A[i*M + k] * B[k*M + j];
			}
		}
	}
	return C;
}

void printMat(vector<int> A, vector<int> B, int N) {
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			if (A[i*N+j] == B[i*N+j]) {
				printf("%-6d", A[i*N+j]);
			} else {
				printf("%2d/%-2d ", A[i*N+j], B[i*N+j]);
			}
		}
		puts("");
	}
}

void printMat(vector<int> M, int N) {
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			printf("%-5d", M[i*N + j]);
		}
		puts("");
	}
}

__global__ void matmul(int* A, int* B, int* C, int M) {
  	unsigned int row, col;
  	row = blockIdx.y * blockDim.y + threadIdx.y;
  	col = blockIdx.x * blockDim.x + threadIdx.x;
 	
	if (row < M && col < M) {
	  	unsigned int idx = col + row * M;
	       	
		// C[i][j] = SUM[k](A[i][k] * B[k][j])	
	  	int k;
	  	for (k = 0; k < M; k++) {
			C[idx] += A[row*M + k] * B[k*M + col];
		}
	}
}

int main(int argc, char **argv)
{	
	if (argc != 4) {
		printf("Multiply MxN matrix by NxK matrix: %s M N K\n", argv[0]);
		return 0;
	}

	int M = atoi(argv[1]);
	int N = atoi(argv[2]);
	int K = atoi(argv[3]);

	int A[M][N], B[N][K];
	for (int i = 0; i < M; i++) {
		for (int j = 0; j < N; j++) {
			A[i][j] = rand() % 10;
		}
	}
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < K; j++) {
			B[i][j] rand() % 10;
		}
	}	

	vector<int> A_data, B_data;
	for (int i = 0; i < M; i++) {
		for (int j = 0; j < N; j++) {
			A_data.push_back(A[i][j]);
		}
	}
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < K; j++) {
			B_data.push_back(B[i][j]);
		}
	}

	vector<int> C_data(A_data.size());

	unsigned int size = A_data.size() * sizeof(int);

	int *dev_A, *dev_B, *dev_C;

	hipMalloc((void**) &dev_A, size);
	hipMalloc((void**) &dev_B, size);
	hipMalloc((void**) &dev_C, size);

	hipMemcpy(dev_A, A_data.data(), size, hipMemcpyHostToDevice);
	hipMemcpy(dev_B, B_data.data(), size, hipMemcpyHostToDevice);

	dim3 gridSize(1,1), blockSize(N,N);

	if (N > 32) {
		gridSize.x = gridSize.y = ceil((double)N/32);
		blockSize.x = blockSize.y = 32;
	}

	matmul<<<gridSize,blockSize>>>(dev_A, dev_B, dev_C, N);
	hipDeviceSynchronize();

	hipMemcpy(C_data.data(), dev_C, size, hipMemcpyDeviceToHost);
	
	if (C_data != A_data) {
		printMat(A_data, C_data, N);
	}

	assert(C_data == A_data);
}

