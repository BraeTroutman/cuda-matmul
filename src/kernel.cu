#include "hip/hip_runtime.h"
#include <vector>
#include <hip/hip_runtime.h>

#include "kernel.h"

__global__ void kernel(int* A, int* B, int* C, int M, int N, int K) {
  	unsigned int row, col;
  	row = blockIdx.y * blockDim.y + threadIdx.y;
  	col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < M && col < K) {
	  	unsigned int idx = col + row * K;

		// C[i][j] = SUM[k](A[i][k] * B[k][j])
		C[idx] = 0;
	  	for (int i = 0; i < N; i++) {
			C[idx] += A[row*N + i] * B[i*K + col];
		}
	}
}

std::vector<int> cudaMatmul(std::vector<int> &A, std::vector<int>& B, int M, int N, int K) {
	std::vector<int> C;

	int *d_A, *d_B, *d_C;

	hipMalloc((void**) &d_A, sizeof(int)*M*N);
	hipMalloc((void**) &d_B, sizeof(int)*N*K);
	hipMalloc((void**) &d_C, sizeof(int)*M*K);

	hipMemcpy(d_A, A.data(), sizeof(int)*M*N, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B.data(), sizeof(int)*N*K, hipMemcpyHostToDevice);
	
	dim3 gridSize(1,1), blockSize(K,M);

	if (M > 32 || K > 32) {
		gridSize.x = ceil((double)K/32);
		gridSize.y = ceil((double)M/32);
		blockSize.x = blockSize.y = 32;
	}

	kernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, M, N, K);
	hipDeviceSynchronize();

	hipMemcpy(C.data(), d_C, sizeof(int)*M*K, hipMemcpyDeviceToHost);

	return C;
}


